#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float devData;

__global__ void checkGlobalVariable()
{
    printf("Device: the value of the global variable is %f\n", devData);
    devData += 2.0f;
}

int main(void)
{
    float value = 3.14f;
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float)));
    printf("Host: copied %f to the global variable\n", value);

    // invoke the kernel
    checkGlobalVariable<<<1,1>>>();

    CHECK(hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float)));
    printf("Host: the value changed by the kernel to %f\n", value);

    CHECK(hipDeviceReset());
    return 0;
}
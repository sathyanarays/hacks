#include "hip/hip_runtime.h"
#include "common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define LOG 0

int cpuRecursiveReduce(int *data, int const size)
{
    if (size == 1) return data[0];

    int const stride = size / 2;

    for (int i=0; i<stride; i++)
    {
        data[i] += data[i+stride];
    }

    return cpuRecursiveReduce(data, stride);
}

__global__ void gpuRecursiveReduce(int *g_idata, int *g_odata, unsigned int isize)
{
    unsigned int tid = threadIdx.x;

    int *idata = g_idata + blockIdx.x * blockDim.x;
    int *odata = &g_odata[blockIdx.x];

    if (isize == 2 && tid == 0)
    {
        g_odata[blockIdx.x] = idata[0] + idata[1];
        return;
    }

    int istride = isize >> 1;

    if(istride > 1 && tid < istride)
    {
        idata[tid] += idata[tid + istride];
    }

    __syncthreads();

    if(tid == 0)
    {
        gpuRecursiveReduce<<<1, istride>>>(idata, odata, istride);

        hipDeviceSynchronize();
    }

    __syncthreads();
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0, gpu_sum;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    bool bResult = false;

    // set up execution configuration
    int nblock  = 2048;
    int nthread = 512;   // initial block size

    if(argc > 1)
    {
        nblock = atoi(argv[1]);   // block size from command line argument
    }

    if(argc > 2)
    {
        nthread = atoi(argv[2]);   // block size from command line argument
    }

    int size = nblock * nthread; // total number of elements to reduceNeighbored

    dim3 block (nthread, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("array %d grid %d block %d\n", size, grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(int);
    int *h_idata = (int *) malloc(bytes);
    int *h_odata = (int *) malloc(grid.x * sizeof(int));
    int *tmp     = (int *) malloc(bytes);

    // initialize the array
    for (int i = 0; i < size; i++)
    {
        h_idata[i] = (int)( rand() & 0xFF );
        h_idata[i] = 1;
    }

    memcpy (tmp, h_idata, bytes);

    // allocate device memory
    int *d_idata = NULL;
    int *d_odata = NULL;
    CHECK(hipMalloc((void **) &d_idata, bytes));
    CHECK(hipMalloc((void **) &d_odata, grid.x * sizeof(int)));

    double iStart, iElaps;

    // cpu recursive reduction
    iStart = seconds();
    int cpu_sum = cpuRecursiveReduce (tmp, size);
    iElaps = seconds() - iStart;
    printf("cpu reduce\t\telapsed %f sec cpu_sum: %d\n", iElaps, cpu_sum);

    // gpu reduceNeighbored
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    iStart = seconds();
    //reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    iElaps = seconds() - iStart;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("gpu Neighbored\t\telapsed %f sec gpu_sum: %d <<<grid %d block "
           "%d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    // gpu nested reduce kernel
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    iStart = seconds();
    gpuRecursiveReduce<<<grid, block>>>(d_idata, d_odata, block.x);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    iElaps = seconds() - iStart;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("gpu nested\t\telapsed %f sec gpu_sum: %d <<<grid %d block %d>>>\n",
           iElaps, gpu_sum, grid.x, block.x);

    // free host memory
    free(h_idata);
    free(h_odata);

    // free device memory
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));

    // reset device
    CHECK(hipDeviceReset());

    // check the results
    bResult = (gpu_sum == cpu_sum);

    if(!bResult) printf("Test failed!\n");

    return EXIT_SUCCESS;
}
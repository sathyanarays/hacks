#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <unistd.h>


int main (void){
    hipError_t cudaStat;    
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int i, j;
    float* v1;
    float* v2;
    float* out;
    float* buffer1;
    float* buffer2;
    float* buffer3;
    
    buffer1 = (float *)malloc(5 * sizeof(float));    
    buffer2 = (float *)malloc(5 * sizeof(float));    
    buffer3 = (float *)malloc(5 * sizeof(float));    
    hipMalloc ((void**)&v1, 5 * sizeof (*v1));
    hipMalloc ((void**)&v2, 5 * sizeof (*v1));
    hipMalloc ((void**)&out, 5 * sizeof (*v1));
    
    buffer1[0] = 2.0;
    buffer1[1] = 1.0;
    buffer1[2] = 1.0;
    buffer1[3] = 1.0;
    buffer1[4] = 1.0;

    hipMemcpy(v1, buffer1, 5 * sizeof(float), hipMemcpyHostToDevice);

    buffer2[0] = 2.0;
    buffer2[1] = 2.0;
    buffer2[2] = 3.0;
    buffer2[3] = 0.5;
    buffer2[4] = 1.0;

    hipMemcpy(v2, buffer2, 5 * sizeof(float), hipMemcpyHostToDevice);
    
    printf("Here\n");
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    hipblasSdot(handle,5,v1,1,v2,1,out);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("Dot product failed\n");
        return EXIT_FAILURE;
    }

    printf("Here\n");
    hipMemcpy(buffer3, out, 5 * sizeof(float), hipMemcpyDeviceToHost);
    
    for(i=0;i<5;i++){
        printf("%f ", buffer3[i]);
    }
    
    sleep(1000);

    hipblasDestroy(handle);
}